#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N 10000
#define BLOCK_SIZE 256
__global__ void matrix_add(double *a, double *b, double *c, size_t pitch, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n) {
        double *rowA = (double *)((char *)a + row * pitch);
        double *rowB = (double *)((char *)b + row * pitch);
        double *rowC = (double *)((char *)c + row * pitch);

        for (int col = 0; col < n; col++) {
            rowC[col] = rowA[col] + rowB[col];
        }
    }
}

int main() {
    size_t pitch;
    

    double **h_a = (double **)malloc(N * sizeof(double *));
    double **h_b = (double **)malloc(N * sizeof(double *));
    double **h_c = (double **)malloc(N * sizeof(double *));
    
    for (int i = 0; i < N; i++) {
        h_a[i] = (double *)malloc(N * sizeof(double));
        h_b[i] = (double *)malloc(N * sizeof(double));
        h_c[i] = (double *)malloc(N * sizeof(double));
    }

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_a[i][j] = (double)(rand() % 1000 + 1);
            h_b[i][j] = (double)(rand() % 1000 + 1);
            h_c[i][j] = 0.0;
        }
    }

    double *d_a, *d_b, *d_c;
    hipMallocPitch((void **)&d_a, &pitch, N * sizeof(double), N);
    hipMallocPitch((void **)&d_b, &pitch, N * sizeof(double), N);
    hipMallocPitch((void **)&d_c, &pitch, N * sizeof(double), N);

    for (int i = 0; i < N; i++) {
        hipMemcpy2D((char *)d_a + i * pitch, pitch, h_a[i], N * sizeof(double), N * sizeof(double), 1, hipMemcpyHostToDevice);
        hipMemcpy2D((char *)d_b + i * pitch, pitch, h_b[i], N * sizeof(double), N * sizeof(double), 1, hipMemcpyHostToDevice);
    }

    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    double start_time = clock();

    matrix_add<<<numBlocks, BLOCK_SIZE>>>(d_a, d_b, d_c, pitch, N);
    hipDeviceSynchronize();

    double end_time = clock();
    printf("Parallel Matrix Addition Time: %f seconds\n", (end_time - start_time) / CLOCKS_PER_SEC);

    for (int i = 0; i < N; i++) {
        hipMemcpy2D(h_c[i], N * sizeof(double), (char *)d_c + i * pitch, pitch, N * sizeof(double), 1, hipMemcpyDeviceToHost);
    }

    for (int i = 0; i < N; i++) {
        free(h_a[i]);
        free(h_b[i]);
        free(h_c[i]);
    }
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
